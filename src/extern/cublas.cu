#include <hipblas.h>
#include <hip/hip_runtime.h>

extern "C" void cublas_multiply_f(const float* A, const float* B, float* C,
                                  int M, int N, int K) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alpha = 1.0f, beta = 0.0f;

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M*K*sizeof(float));
    hipMalloc(&d_B, K*N*sizeof(float));
    hipMalloc(&d_C, M*N*sizeof(float));

    hipMemcpy(d_A, A, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K*N*sizeof(float), hipMemcpyHostToDevice);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, M, K, &alpha,
                d_B, N,
                d_A, K,
                &beta,
                d_C, N);

    hipMemcpy(C, d_C, M*N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipblasDestroy(handle);
}